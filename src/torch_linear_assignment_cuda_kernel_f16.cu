#include "hip/hip_runtime.h"
/*
  Implementation is based on the algorithm presented in pages 1685-1686 of:

  DF Crouse. On implementing 2D rectangular assignment algorithms.
    IEEE Transactions on Aerospace and Electronic Systems
    52(4):1679-1696, August 2016
    doi: 10.1109/TAES.2016.140952
*/

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

typedef unsigned char uint8_t;

// Optimized for modern GPUs (Ampere+)
constexpr int BLOCK_SIZE = 256;  // Better for half-precision operations

template <typename uint8_t>
__device__ __forceinline__
void array_fill(uint8_t* start, uint8_t* stop, uint8_t value) {
    for (; start < stop; ++start) {
        *start = value;
    }
}

__device__ __forceinline__
int prune_costs_half(int nr, int nc, __half* cost) {
    __half padVal = cost[nc - 1];
    for (int c = 0; c < nc; c++) {
        if (__hne(cost[c], padVal)) continue;
        
        bool allPad = true;
        for (int r = 0; r < nr; r++) {
            if (__hne(cost[r * nr + c], padVal)) {
                allPad = false;
                break;
            }
        }
        if (allPad) return c;
    }
    return nc;
}

__device__ __forceinline__
int augmenting_path_half(int nr, int nc, int i,
                        __half* cost, __half* u, __half* v,
                        int* path, int* row4col,
                        __half* shortestPathCosts,
                        uint8_t* SR, uint8_t* SC,
                        int* remaining,
                        __half* p_minVal,
                        __half infinity,
                        int limit) {
    __half minVal = __float2half(0.0f);
    int num_remaining = min(nc, limit);

    for (int it = 0; it < limit; ++it) {
        SC[it] = 0;
        remaining[it] = limit - it - 1;
        shortestPathCosts[it] = infinity;
    }

    array_fill(SR, SR + nr, (uint8_t)0);

    int sink = -1;
    while (sink == -1) {
        int index = -1;
        __half lowest = infinity;
        SR[i] = 1;

        __half* cost_row = cost + i * nc;
        __half base_r = __hsub(minVal, u[i]);
        
        for (int it = 0; it < num_remaining; it++) {
            int j = remaining[it];
            __half r = __hadd(base_r, __hsub(cost_row[j], v[j]));
            
            if (__hlt(r, shortestPathCosts[j])) {
                path[j] = i;
                shortestPathCosts[j] = r;
            }
            
            if (__hlt(shortestPathCosts[j], lowest) || 
                (__heq(shortestPathCosts[j], lowest) && row4col[j] == -1)) {
                lowest = shortestPathCosts[j];
                index = it;
            }
        }

        minVal = lowest;
        if (__hisinf(minVal)) {
            return -1;
        }

        int j = remaining[index];
        if (row4col[j] == -1) {
            sink = j;
        } else {
            i = row4col[j];
            if (SR[i]) {
                return -1;  // Cycle detected
            }
        }
        SC[j] = 1;
        remaining[index] = remaining[--num_remaining];
    }
    *p_minVal = minVal;
    return sink;
}

__global__
void solve_kernel_half(int bs, int nr, int nc,
                      __half* cost,
                      __half* u, __half* v,
                      __half* shortestPathCosts,
                      int* path, int* col4row, int* row4col,
                      uint8_t* SR, uint8_t* SC,
                      int* remaining,
                      __half infinity,
                      int* limits) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= bs) return;

    int limit = limits[i];
    __half minVal;
    
    for (int curRow = 0; curRow < nr; ++curRow) {
        int sink = augmenting_path_half(nr, nc, curRow, 
                                      cost + i * nr * nc,
                                      u + i * nr,
                                      v + i * nc,
                                      path + i * nc,
                                      row4col + i * nc,
                                      shortestPathCosts + i * nc,
                                      SR + i * nr,
                                      SC + i * nc,
                                      remaining + i * nc,
                                      &minVal,
                                      infinity,
                                      limit);

        if (sink < 0) continue;

        u[i * nr + curRow] = __hadd(u[i * nr + curRow], minVal);
        
        for (int r = 0; r < nr; r++) {
            if (SR[i * nr + r] && r != curRow) {
                __half update = __hsub(minVal, shortestPathCosts[i * nc + col4row[i * nr + r]]);
                u[i * nr + r] = __hadd(u[i * nr + r], update);
            }
        }

        for (int c = 0; c < limit; c++) {
            if (SC[i * nc + c]) {
                __half update = __hsub(minVal, shortestPathCosts[i * nc + c]);
                v[i * nc + c] = __hsub(v[i * nc + c], update);
            }
        }

        int j = sink;
        int iterations = 0;
        while (iterations++ < limit) {
            int r = path[i * nc + j];
            if (r == -1) break;
            
            row4col[i * nc + j] = r;
            int temp = j;
            j = col4row[i * nr + r];
            col4row[i * nr + r] = temp;
        }
    }
}

void solve_half_batch(torch::Tensor cost, torch::Tensor col4row, torch::Tensor row4col) {
    auto sizes = cost.sizes();
    int bs = sizes[0], nr = sizes[1], nc = sizes[2];
    int device_index = cost.device().index();
    
    hipSetDevice(device_index);
    auto stream = at::cuda::getCurrentCUDAStream(device_index);

    // Create tensors with half-precision
    auto options = torch::TensorOptions()
        .dtype(torch::kFloat16)
        .device(torch::kCUDA, device_index);
    
    torch::Tensor u = torch::zeros({bs * nr}, options);
    torch::Tensor v = torch::zeros({bs * nc}, options);
    torch::Tensor shortestPathCosts = torch::empty({bs * nc}, options);
    
    // Infinity representation for half-precision
    __half infinity;
    *reinterpret_cast<unsigned short*>(&infinity) = 0x7C00;  // FP16 infinity
    
    // Integer tensors
    auto int_options = torch::TensorOptions()
        .dtype(torch::kInt32)
        .device(torch::kCUDA, device_index);
        
    torch::Tensor path = torch::full({bs * nc}, -1, int_options);
    torch::Tensor remaining = torch::empty({bs * nc}, int_options);
    torch::Tensor limits = torch::full({bs}, nc, int_options);

    // Byte tensors
    auto byte_options = torch::TensorOptions()
        .dtype(torch::kUInt8)
        .device(torch::kCUDA, device_index);
        
    torch::Tensor SR = torch::empty({bs * nr}, byte_options);
    torch::Tensor SC = torch::empty({bs * nc}, byte_options);

    // Launch kernel
    int grid_size = (bs + BLOCK_SIZE - 1) / BLOCK_SIZE;
    solve_kernel_half<<<grid_size, BLOCK_SIZE, 0, stream.stream()>>>(
        bs, nr, nc,
        cost.data_ptr<__half>(),
        u.data_ptr<__half>(),
        v.data_ptr<__half>(),
        shortestPathCosts.data_ptr<__half>(),
        path.data_ptr<int>(),
        col4row.data_ptr<int>(),
        row4col.data_ptr<int>(),
        SR.data_ptr<uint8_t>(),
        SC.data_ptr<uint8_t>(),
        remaining.data_ptr<int>(),
        infinity,
        limits.data_ptr<int>()
    );

    // Error checking
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }
}
 
std::vector<torch::Tensor> solve_half(torch::Tensor cost) {
    auto sizes = cost.sizes();  
    auto device = cost.device();
    auto options = torch::TensorOptions()
      .dtype(torch::kInt)
      .device(device.type(), device.index());
    torch::Tensor col4row = torch::full({sizes[0], sizes[1]}, -1, options);
    torch::Tensor row4col = torch::full({sizes[0], sizes[2]}, -1, options);
  
    // If sizes[2] is zero, then sizes[1] is also zero.
    if (sizes[0] * sizes[1] == 0) {
      return {col4row, row4col};
    }
  
    AT_DISPATCH_FLOATING_TYPES(cost.scalar_type(), "solve_half_batch", [&] {
        solve_half_batch<scalar_t>(
          cost.scalar_type(),
          device.index(),
          sizes[0], sizes[1], sizes[2],
          cost.data<scalar_t>(),
          col4row.data<int>(),
          row4col.data<int>());
    });
    return {col4row, row4col};
  }


